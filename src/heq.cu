
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>

#define TIMER_CREATE(t)           \
  hipEvent_t t##_start, t##_end; \
  hipEventCreate(&t##_start);    \
  hipEventCreate(&t##_end);

#define TIMER_START(t)        \
  hipEventRecord(t##_start); \
  hipEventSynchronize(t##_start);

#define TIMER_END(t)                            \
  hipEventRecord(t##_end);                     \
  hipEventSynchronize(t##_end);                \
  hipEventElapsedTime(&t, t##_start, t##_end); \
  hipEventDestroy(t##_start);                  \
  hipEventDestroy(t##_end);

#define TILE_SIZE 16
#define BLOCK_SIZE_1D 256
#define NUM_BINS 256

#define CUDA_TIMING
#define DEBUG

#define WARP_SIZE 32
#define R 9

#define INTDIVIDE_CEILING(i, N) (((i) + (N)-1) / (N))

unsigned char *input_gpu;
unsigned char *output_gpu;

double CLOCK()
{
  struct timespec t;
  clock_gettime(CLOCK_MONOTONIC, &t);
  return (t.tv_sec * 1000) + (t.tv_nsec * 1e-6);
}

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    exit(-1);
  }
#endif
  return result;
}

// Add GPU kernel and functions
// HERE!!!
inline __device__ void
incPrivatized32Element(unsigned char pixval)
{
  extern __shared__ unsigned int privHist[];
  const int blockDimx = 64;
  unsigned int increment = 1 << 8 * (pixval & 3);
  int index = pixval >> 2;
  privHist[index * blockDimx + threadIdx.x] += increment;
}

template <bool bClear>
__device__ void
merge64HistogramsToOutput(unsigned int *histogram)
{
  extern __shared__ unsigned int privHist[];

  unsigned int sum02 = 0;
  unsigned int sum13 = 0;
  for (int i = 0; i < 64; i++)
  {
    int index = (i + threadIdx.x) & 63;
    unsigned int myValue = privHist[threadIdx.x * 64 + index];
    if (bClear)
      privHist[threadIdx.x * 64 + index] = 0;
    sum02 += myValue & 0xff00ff;
    myValue >>= 8;
    sum13 += myValue & 0xff00ff;
  }

  atomicAdd(&histogram[threadIdx.x * 4 + 0], sum02 & 0xffff);
  sum02 >>= 16;
  atomicAdd(&histogram[threadIdx.x * 4 + 2], sum02);

  atomicAdd(&histogram[threadIdx.x * 4 + 1], sum13 & 0xffff);
  sum13 >>= 16;
  atomicAdd(&histogram[threadIdx.x * 4 + 3], sum13);
}

__global__ void
histogram1DPerThread4x64(
    unsigned int *histogram,
    const unsigned char *input, int N)
{
  extern __shared__ unsigned int privHist[];
  const int blockDimx = 64;

  if (blockDim.x != blockDimx)
    return;

  for (int i = threadIdx.x;
       i < 64 * blockDimx;
       i += blockDimx)
  {
    privHist[i] = 0;
  }
  __syncthreads();
  int cIterations = 0;
  for (int i = blockIdx.x * blockDimx + threadIdx.x;
       i < N / 4;
       i += blockDimx * gridDim.x)
  {
    unsigned int value = ((unsigned int *)input)[i];
    incPrivatized32Element(value & 0xff);
    value >>= 8;
    incPrivatized32Element(value & 0xff);
    value >>= 8;
    incPrivatized32Element(value & 0xff);
    value >>= 8;
    incPrivatized32Element(value);
    cIterations += 1;
    if (false && cIterations >= 252 / 4)
    {
      cIterations = 0;
      __syncthreads();
      merge64HistogramsToOutput<true>(histogram);
    }
  }
  __syncthreads();

  merge64HistogramsToOutput<false>(histogram);
}

__global__ void
histogram1DPerBlock(
    unsigned int *pHist,
    const unsigned char *base, int N)
{
  __shared__ int sHist[256];
  for (int i = threadIdx.x;
       i < 256;
       i += blockDim.x)
  {
    sHist[i] = 0;
  }
  __syncthreads();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < N;
       i += blockDim.x * gridDim.x)
  {
    unsigned int value = ((unsigned int *)base)[i];

    atomicAdd(&sHist[value & 0xff], 1);
    value >>= 8;
    atomicAdd(&sHist[value & 0xff], 1);
    value >>= 8;
    atomicAdd(&sHist[value & 0xff], 1);
    value >>= 8;
    atomicAdd(&sHist[value], 1);
  }
  __syncthreads();
  for (int i = threadIdx.x;
       i < 256;
       i += blockDim.x)
  {
    atomicAdd(&pHist[i], sHist[i]);
  }
}

__global__ void
histogram1DPerGrid(
    unsigned int *pHist,
    const unsigned char *base, int N)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < N;
       i += blockDim.x * gridDim.x)
  {
    unsigned int value = ((unsigned int *)base)[i];
    atomicAdd(&pHist[value & 0xff], 1);
    value >>= 8;
    atomicAdd(&pHist[value & 0xff], 1);
    value >>= 8;
    atomicAdd(&pHist[value & 0xff], 1);
    value >>= 8;
    atomicAdd(&pHist[value], 1);
  }
}

__global__ void kernel(unsigned char *input, unsigned int *output_cdf,
                       unsigned int im_size, unsigned int *cdf_min)
{

  int location = blockIdx.x * blockDim.x + threadIdx.x;
  input[location] = float(output_cdf[input[location]] - *cdf_min) / float(im_size / 64 - *cdf_min) * (NUM_BINS - 1);
}

__global__ void get_histogram(unsigned char *input,
                              unsigned int *output_histogram)
{
  if (!(threadIdx.x & 63))
  {

    int location = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&(output_histogram[input[location]]), 1);
  }

  __syncthreads();
}

__global__ void get_cdf_prefixSum(unsigned int *histogram)
{
  int tid = threadIdx.x;

  //USE SHARED MEMORY - COMON WE ARE EXPERIENCED PROGRAMMERS
  __shared__ int Cache[256];
  Cache[tid] = histogram[tid];
  __syncthreads();
  int space = 1;

  //BEGIN
  for (int i = 0; i < 8; i++)
  {
    int temp = Cache[tid];
    int neighbor = 0;
    if ((tid - space) >= 0)
    {
      neighbor = Cache[tid - space];
    }
    __syncthreads(); //AFTER LOADING

    if (tid < space)
    {
      //DO NOTHING
    }
    else
    {
      Cache[tid] = temp + neighbor;
    }

    space = space * 2;
    __syncthreads();
  }

  //REWRITE RESULTS TO MAIN MEMORY
  histogram[tid] = Cache[tid];
}

__global__ void reductionMin(unsigned int *sdata, unsigned int *results, int n)
{
  // extern __shared__ int sdata[];
  unsigned int tx = threadIdx.x;

  // block-wide reduction
  for (unsigned int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
  {
    __syncthreads();
    if (tx < offset)
    {
      if (sdata[tx + offset] < sdata[tx] || sdata[tx] == 0)
        sdata[tx] = sdata[tx + offset];
    }
  }
  // finally, thread 0 writes the result
  if (threadIdx.x == 0)
  {
    // the result is per-block
    *results = sdata[0];
  }
}

__global__ void kernel_warmup(unsigned char *input,
                              unsigned char *output)
{

  int x = blockIdx.x * TILE_SIZE + threadIdx.x;
  int y = blockIdx.y * TILE_SIZE + threadIdx.y;

  int location = y * TILE_SIZE * gridDim.x + x;
  output[location] = x % 255;
}

void histogram_gpu(unsigned char *data,
                   unsigned int height,
                   unsigned int width)
{

  int gridXSize = 1 + ((width - 1) / TILE_SIZE);
  int gridYSize = 1 + ((height - 1) / TILE_SIZE);
  int gridSize_1D = 1 + (NUM_BINS - 1) / BLOCK_SIZE_1D;

  int gridSize1D_2D = 1 + ((width * height - 1) / BLOCK_SIZE_1D);

  int XSize = gridXSize * TILE_SIZE;
  int YSize = gridYSize * TILE_SIZE;

  // Both are the same size (CPU/GPU).
  int size = XSize * YSize;

  // CPU
  unsigned int *cdf_gpu = new unsigned int[NUM_BINS];

  // GPU
  unsigned int *histogram;
  unsigned int *cdf_min;

  // bool bPeriodicMerge = false;
  // dim3 threads(16, 4, 1);
  // int numthreads = threads.x * threads.y;
  // int numblocks = bPeriodicMerge ? 256 : INTDIVIDE_CEILING(size, numthreads * (255 / 4));

  // Allocate arrays in GPU memory
  checkCuda(hipMalloc((void **)&input_gpu, size * sizeof(unsigned char)));
  checkCuda(hipMalloc((void **)&histogram, NUM_BINS * sizeof(unsigned int)));
  checkCuda(hipMalloc((void **)&cdf_min, sizeof(unsigned int)));

  checkCuda(hipMemset(histogram, 0, NUM_BINS * sizeof(unsigned int)));
  checkCuda(hipMemset(cdf_min, 0, sizeof(unsigned int)));

  // Copy data to GPU
  checkCuda(hipMemcpy(input_gpu,
                       data,
                       size * sizeof(char),
                       hipMemcpyHostToDevice));

  checkCuda(hipDeviceSynchronize());

  // Execute algorithm
  dim3 dimGrid2D(gridXSize, gridYSize);
  dim3 dimBlock2D(TILE_SIZE, TILE_SIZE);

  dim3 dimGrid1D(gridSize_1D);
  dim3 dimBlock1D(BLOCK_SIZE_1D);

  dim3 dimGrid1D_2D(gridSize1D_2D);
  dim3 dimBlock1D_2D(BLOCK_SIZE_1D);

// Kernel Call
#if defined(CUDA_TIMING)
  float Ktime;
  TIMER_CREATE(Ktime);
  TIMER_START(Ktime);
#endif
  //histogram_generation<<<5,256>>>(histogram, input_gpu, width*height);
  //histogram256Kernel<<<gridXSize*gridYSize, 256>>>(histogram, input_gpu, width*height);
  // histogram1DPerThread4x64<<<numblocks, numthreads, numthreads * 256>>>(histogram, input_gpu, size);
  // histogram1DPerBlock<<<400,256/*threads.x*threads.y*/>>>( histogram, input_gpu, width * height / 4);
  // histogram1DPerGrid<<<400,256/*threads.x*threads.y*/>>>( histogram, input_gpu, width * height / 4);
  // get_cdf<<<dimGrid1D, dimBlock1D>>>(histogram, histogram, NUM_BINS);
  get_histogram<<<dimGrid1D_2D, dimBlock1D_2D>>>(input_gpu, histogram);
  get_cdf_prefixSum<<<1, 256>>>(histogram);

  checkCuda(hipPeekAtLastError());
  checkCuda(hipDeviceSynchronize());

  reductionMin<<<1, 256>>>(histogram, cdf_min, 256);
  kernel<<<dimGrid1D_2D, dimBlock1D_2D>>>(input_gpu, histogram, width * height, cdf_min);
  checkCuda(hipPeekAtLastError());
  checkCuda(hipDeviceSynchronize());

#if defined(CUDA_TIMING)
  TIMER_END(Ktime);
  printf("Kernel Execution Time: %f ms\n", Ktime);
#endif

  checkCuda(hipMemcpy(data,
                       input_gpu,
                       size * sizeof(unsigned char),
                       hipMemcpyDeviceToHost));

  checkCuda(hipFree(histogram));
  checkCuda(hipFree(cdf_min));
  checkCuda(hipFree(input_gpu));
}

void histogram_gpu_warmup(unsigned char *data,
                          unsigned int height,
                          unsigned int width)
{

  int gridXSize = 1 + ((width - 1) / TILE_SIZE);
  int gridYSize = 1 + ((height - 1) / TILE_SIZE);

  int XSize = gridXSize * TILE_SIZE;
  int YSize = gridYSize * TILE_SIZE;

  // Both are the same size (CPU/GPU).
  int size = XSize * YSize;

  // Allocate arrays in GPU memory
  checkCuda(hipMalloc((void **)&input_gpu, size * sizeof(unsigned char)));
  checkCuda(hipMalloc((void **)&output_gpu, size * sizeof(unsigned char)));

  checkCuda(hipMemset(output_gpu, 0, size * sizeof(unsigned char)));

  // Copy data to GPU
  checkCuda(hipMemcpy(input_gpu,
                       data,
                       size * sizeof(char),
                       hipMemcpyHostToDevice));

  checkCuda(hipDeviceSynchronize());

  // Execute algorithm

  dim3 dimGrid(gridXSize, gridYSize);
  dim3 dimBlock(TILE_SIZE, TILE_SIZE);

  kernel_warmup<<<dimGrid, dimBlock>>>(input_gpu,
                                       output_gpu);

  checkCuda(hipDeviceSynchronize());

  // Retrieve results from the GPU
  checkCuda(hipMemcpy(data,
                       output_gpu,
                       size * sizeof(unsigned char),
                       hipMemcpyDeviceToHost));

  // Free resources and end the program
  checkCuda(hipFree(output_gpu));
  checkCuda(hipFree(input_gpu));
}
